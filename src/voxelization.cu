#include "hip/hip_runtime.h"

#include "voxelization.h"
#include <glm/glm.hpp>
#include <GL/glut.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include <voxelpipe/voxelpipe.h>

#include "timingUtils.h"

voxelpipe::FRContext<log_N, log_T>*  context;
bool first_time = true;

struct ColorShader
{
  glm::vec3* texture;
  int tex_width;
  int tex_height;
  float* texcoord;
  int texcoord_size;

  __device__ float shade(
    const int tri_id,
    const float4 v0,
    const float4 v1,
    const float4 v2,
    const float3 n,
    const float  bary0,
    const float  bary1,
    const int3   xyz) const
  {
    //If there is no texture, just return green
    if (tex_width == 0) {
      return __int_as_float((255 << 8) + (127 << 24));
    }

    //If there are no texcoordinates, just return the first value in the texture
    if (texcoord_size == 0) {
      int r = (int)(texture[0].r * 255.0);
      int g = (int)(texture[0].g * 255.0);
      int b = (int)(texture[0].b * 255.0);
      return __int_as_float(r+(g << 8) + (b << 16) + (127 << 24));
    }

    //Get the texture coordinates from the triangle id
    int t1_x = texcoord[6 * tri_id] * tex_width;
    int t1_y = texcoord[6 * tri_id + 1] * tex_height;
    int t2_x = texcoord[6 * tri_id + 2] * tex_width;
    int t2_y = texcoord[6 * tri_id + 3] * tex_height;
    int t3_x = texcoord[6 * tri_id + 4] * tex_width;
    int t3_y = texcoord[6 * tri_id + 5] * tex_height;

    //Get the colors from the texture at these vertices
    glm::vec3 c1 = texture[t1_y * tex_width + t1_x];
    glm::vec3 c2 = texture[t2_y * tex_width + t2_x];
    glm::vec3 c3 = texture[t3_y * tex_width + t3_x];

    //TODO: Interpolate using barycentric coordinates
    glm::vec3 color = c1;

    //Compute rgb components
    int r = (int) (clamp(color.r, 0.0f, 1.0f) * 255.0f);
    int g = (int) (clamp(color.g, 0.0f, 1.0f) * 255.0f);
    int b = (int) (clamp(color.b, 0.0f, 1.0f) * 255.0f);

    //Compact
    int val = r + (g << 8) + (b << 16) + (127 << 24);

    return __int_as_float(val);
  }
};

__global__ void getOccupiedVoxels(void* fb, int M, int T, int* voxels) {
  int T3 = T*T*T;
  int M3 = M*M*M;

  int pix_num = (blockIdx.x * 256 % T3) + threadIdx.x;
  int tile_num = blockIdx.x * 256 / T3;

  if (pix_num < T3 && tile_num < M3) {
    //TODO: Is there any benefit in making this shared?
    float* tile;

    bool is_occupied;
    tile = (float*)fb + tile_num*T3;
    int alpha = __float_as_int(tile[pix_num]) >> 24;
    is_occupied = alpha > 0;

    if (is_occupied) {
      voxels[tile_num*T3 + pix_num] = tile_num*T3 + pix_num;
    } else {
      voxels[tile_num*T3 + pix_num] = -1;
    }
  }

}

//Thrust predicate for removal of empty voxels
struct check_voxel {
  __host__ __device__
    bool operator() (const int& c) {
    return (c != -1);
  }
};

__global__ void extractValues(void* fb, int* voxels, int num_voxels, int* values) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < num_voxels) {
    //TODO: Make this support other storage_type's besides int32
    float* tile = (float*)fb;
    values[index] = __float_as_int(tile[voxels[index]]);
  }
}

__global__ void createCubeMesh(int* voxels, int* values, int M, int T, float3 bbox0, float3 t_d, float3 p_d, float scale_factor, int num_voxels, float* cube_vbo, 
                                int cube_vbosize, int* cube_ibo, int cube_ibosize, float* cube_nbo, float* out_vbo, int* out_ibo, float* out_nbo, float* out_cbo) {

  //Get the index for the thread
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if (idx < num_voxels) {

    int vbo_offset = idx * cube_vbosize;
    int ibo_offset = idx * cube_ibosize;
    float3 center = getCenterFromIndex(voxels[idx], M, T, bbox0, t_d, p_d);
    int color = values[idx];

    for (int i = 0; i < cube_vbosize; i++) {
      if (i % 3 == 0) {
        out_vbo[vbo_offset + i] = cube_vbo[i] * scale_factor + center.x;
        out_cbo[vbo_offset + i] = (float)((color & 0xFF)/255.0);
      } else if (i % 3 == 1) {
        out_vbo[vbo_offset + i] = cube_vbo[i] * scale_factor + center.y;
        out_cbo[vbo_offset + i] = (float)(((color >> 8) & 0xFF)/255.0);
      } else {
        out_vbo[vbo_offset + i] = cube_vbo[i] * scale_factor + center.z;
        out_cbo[vbo_offset + i] = (float)(((color >> 16) & 0xFF)/255.0);
      }
      out_nbo[vbo_offset + i] = cube_nbo[i];
    }

    for (int i = 0; i < cube_ibosize; i++) {
      out_ibo[ibo_offset + i] = cube_ibo[i] + ibo_offset;
    }

  }

}

__host__ int voxelizeMesh(Mesh &m_in, bmp_texture* h_tex, int* d_voxels, int* d_values) {

  //Initialize sizes
  const int n_triangles = m_in.ibosize / 3;
  const int n_vertices = m_in.vbosize / 3;

  //Create host vectors
  thrust::host_vector<int4> h_triangles(n_triangles);
  thrust::host_vector<float4> h_vertices(n_vertices);

  //Fill in the data
  for (int i = 0; i < n_vertices; i++) {
    h_vertices[i].x = m_in.vbo[i * 3 + 0];
    h_vertices[i].y = m_in.vbo[i * 3 + 1];
    h_vertices[i].z = m_in.vbo[i * 3 + 2];
  }
  for (int i = 0; i < n_triangles; i++) {
    h_triangles[i].x = m_in.ibo[i * 3 + 0];
    h_triangles[i].y = m_in.ibo[i * 3 + 1];
    h_triangles[i].z = m_in.ibo[i * 3 + 2];
  }

  //Copy to device vectors
  thrust::device_vector<int4> d_triangles(h_triangles);
  thrust::device_vector<float4> d_vertices(h_vertices);

  if (first_time) {
    //Create the voxelpipe context
    context = new voxelpipe::FRContext<log_N, log_T>();

    //Reserve data for voxelpipe
    context->reserve(n_triangles, 1024u * 1024u * 16u);
  }
  first_time = false;

  //Initialize the result data on the device
  thrust::device_vector<float>  d_fb(M*M*M * T*T*T);

  //Copy the texture to the device
  glm::vec3 *device_tex = NULL;
  hipMalloc((void**)&device_tex, h_tex->width * h_tex->height *sizeof(glm::vec3));
  hipMemcpy(device_tex, h_tex->data, h_tex->width * h_tex->height *sizeof(glm::vec3), hipMemcpyHostToDevice);

  //Copy the texture coordinates to the device
  float* device_texcoord = NULL;
  hipMalloc((void**)&device_texcoord, m_in.tbosize * sizeof(float));
  hipMemcpy(device_texcoord, m_in.tbo, m_in.tbosize *sizeof(float), hipMemcpyHostToDevice);

  //Create the shader to be used that will write texture colors to voxels
  ColorShader my_shader;
  my_shader.texture = device_tex;
  my_shader.tex_height = h_tex->height;
  my_shader.tex_width = h_tex->width;
  my_shader.texcoord = device_texcoord;
  my_shader.texcoord_size = m_in.tbosize;

  //Perform coarse and fine voxelization
  context->coarse_raster(n_triangles, n_vertices, thrust::raw_pointer_cast(&d_triangles.front()), thrust::raw_pointer_cast(&d_vertices.front()), bbox0, bbox1);
  context->fine_raster< voxelpipe::Float, voxelpipe::FP32S_FORMAT, voxelpipe::THIN_RASTER, voxelpipe::NO_BLENDING, ColorShader >(
    n_triangles, n_vertices, thrust::raw_pointer_cast(&d_triangles.front()), thrust::raw_pointer_cast(&d_vertices.front()), bbox0, bbox1, thrust::raw_pointer_cast(&d_fb.front()), my_shader);

  hipFree(device_tex);
  hipFree(device_texcoord);

  //Get voxel centers
  int numVoxels = N*N*N;
  int* d_vox;
  hipMalloc((void**)&d_vox, numVoxels*sizeof(int));
  getOccupiedVoxels<< <N*N*N, 256 >> >(thrust::raw_pointer_cast(&d_fb.front()), M, T, d_vox);
  hipDeviceSynchronize();

  //Stream Compact voxels to remove the empties
  numVoxels = thrust::copy_if(thrust::device_pointer_cast(d_vox), thrust::device_pointer_cast(d_vox) + numVoxels, thrust::device_pointer_cast(d_voxels), check_voxel()) - thrust::device_pointer_cast(d_voxels);

  std::cout << "Num Voxels: " << numVoxels << std::endl;

  //Extract the values at these indices
  extractValues<<<(numVoxels / 256) + 1, 256 >>>(thrust::raw_pointer_cast(&d_fb.front()), d_voxels, numVoxels, d_values);
  hipDeviceSynchronize();

  hipFree(d_vox);

  return numVoxels;
}

__host__ void extractCubesFromVoxelGrid(int* d_voxels, int numVoxels, int* d_values, Mesh &m_cube, Mesh &m_out) {

  //Move cube data to GPU
  thrust::device_vector<float> d_vbo_cube(m_cube.vbo, m_cube.vbo + m_cube.vbosize);
  thrust::device_vector<int> d_ibo_cube(m_cube.ibo, m_cube.ibo + m_cube.ibosize);
  thrust::device_vector<float> d_nbo_cube(m_cube.nbo, m_cube.nbo + m_cube.nbosize);

  //Create output structs
  float* d_vbo_out;
  int* d_ibo_out;
  float* d_nbo_out;
  float* d_cbo_out;
  hipMalloc((void**)&d_vbo_out, numVoxels * m_cube.vbosize * sizeof(float));
  hipMalloc((void**)&d_ibo_out, numVoxels * m_cube.ibosize * sizeof(int));
  hipMalloc((void**)&d_nbo_out, numVoxels * m_cube.nbosize * sizeof(float));
  hipMalloc((void**)&d_cbo_out, numVoxels * m_cube.nbosize * sizeof(float));

  //Warn if vbo and nbo are not same size on cube
  if (m_cube.vbosize != m_cube.nbosize) {
    std::cout << "ERROR: cube vbo and nbo have different sizes." << std::endl;
    return;
  }

  //Create resulting cube-ized mesh
  createCubeMesh<<<(numVoxels / 256) + 1, 256>>>(d_voxels, d_values, M, T, bbox0, t_d, p_d, vox_size / CUBE_MESH_SCALE, numVoxels, thrust::raw_pointer_cast(&d_vbo_cube.front()),
    m_cube.vbosize, thrust::raw_pointer_cast(&d_ibo_cube.front()), m_cube.ibosize, thrust::raw_pointer_cast(&d_nbo_cube.front()), d_vbo_out, d_ibo_out, d_nbo_out, d_cbo_out);

  //Store output sizes
  m_out.vbosize = numVoxels * m_cube.vbosize;
  m_out.ibosize = numVoxels * m_cube.ibosize;
  m_out.nbosize = numVoxels * m_cube.nbosize;
  m_out.cbosize = m_out.nbosize;

  //Memory allocation for the outputs
  m_out.vbo = (float*)malloc(m_out.vbosize * sizeof(float));
  m_out.ibo = (int*)malloc(m_out.ibosize * sizeof(int));
  m_out.nbo = (float*)malloc(m_out.nbosize * sizeof(float));
  m_out.cbo = (float*)malloc(m_out.cbosize * sizeof(float));

  //Sync here after doing some CPU work
  hipDeviceSynchronize();

  //Copy data back from GPU
  //TODO: Can we avoid this step by making everything run from device-side VBO/IBO/NBO/CBO?
  hipMemcpy(m_out.vbo, d_vbo_out, m_out.vbosize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(m_out.ibo, d_ibo_out, m_out.ibosize*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(m_out.nbo, d_nbo_out, m_out.nbosize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(m_out.cbo, d_cbo_out, m_out.cbosize*sizeof(float), hipMemcpyDeviceToHost);

  ///Free GPU memory
  hipFree(d_vbo_out);
  hipFree(d_ibo_out);
  hipFree(d_nbo_out);
  hipFree(d_cbo_out);

}

__host__ void voxelizeToCubes(Mesh &m_in, bmp_texture* tex, Mesh &m_cube, Mesh &m_out) {
  
  //Voxelize the mesh input
  int numVoxels = N*N*N;
  int* d_voxels;
  int* d_values;
  hipMalloc((void**)&d_voxels, numVoxels*sizeof(int));
  hipMalloc((void**)&d_values, numVoxels*sizeof(int));
  startTiming();
  numVoxels = voxelizeMesh(m_in, tex, d_voxels, d_values);
  std::cout << "Vox Time: " << stopTiming() << std::endl;

  //Extract Cubes from the Voxel Grid
  startTiming();
  extractCubesFromVoxelGrid(d_voxels, numVoxels, d_values, m_cube, m_out);
  std::cout << "Extraction Time: " << stopTiming() << std::endl;

  hipFree(d_voxels);
  hipFree(d_values);
}

