#include "hip/hip_runtime.h"

#include "svo.h"
#include "voxelization.h"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

__global__ void flagNodes(int* voxels, int numVoxels, int* octree, int M, int T, float3 bbox0, float3 t_d, float3 p_d, int tree_depth) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //Don't do anything if its out of bounds
  if (index < numVoxels) {
    float3 center = getCenterFromIndex(voxels[index], M, T, bbox0, t_d, p_d);
    float edge_length = abs(bbox0.x);
    float3 center_depth = make_float3(0.0f, 0.0f, 0.0f);
    int node_idx = 0;
    int this_node;

    //Loop until the specified depth
    for (int i = 0; i < tree_depth; i++) {
      int x = center.x > center_depth.x;
      int y = center.y > center_depth.y;
      int z = center.z > center_depth.z;
      this_node = node_idx + (x + (y << 1) + (z << 2));

      if (i < tree_depth - 1) {
        //The lowest 30 bits are the address
        node_idx = octree[2*this_node] & 0x3FFFFFFF;

        //Update the center depth for the next iteration
        center_depth.x += edge_length / 2 * (x ? 1 : -1);
        center_depth.y += edge_length / 2 * (y ? 1 : -1);
        center_depth.z += edge_length / 2 * (z ? 1 : -1);
      }
      edge_length /= 2.0f;
    }
    octree[2*this_node] = octree[2*this_node] | 0x40000000;
  }

}

__global__ void splitNodes(int* octree, int* numNodes, int poolSize) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //Don't do anything if its out of bounds
  if (index < poolSize) {
    int node = octree[2*index];

    //Split the node if its flagged
    if (node & 0x40000000) {
      //Get a new node tile
      int newNode = atomicAdd(numNodes, 8);

      //Point this node at the new tile
      octree[2 * index] = (octree[2 * index] & 0xC0000000) | (newNode & 0x3FFFFFFF);

      //Initialize new child nodes to 0's
      for (int off = 0; off < 8; off++) {
        octree[2*(newNode + off)] = 0;
        octree[2*(newNode + off) + 1] = 0;
      }
    }
  }

}

__global__ void fillNodes(int* voxels, int numVoxels, int* values, int* octree, int M, int T, float3 bbox0, float3 t_d, float3 p_d) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //Don't do anything if its out of bounds
  if (index < numVoxels) {
    float3 center = getCenterFromIndex(voxels[index], M, T, bbox0, t_d, p_d);
    float edge_length = abs(bbox0.x);
    float3 center_depth = make_float3(0.0f, 0.0f, 0.0f);
    int node_idx = 0;
    int this_node;
    bool has_child = true;

    //Loop until the specified depth
    while (has_child) {
      int x = center.x > center_depth.x;
      int y = center.y > center_depth.y;
      int z = center.z > center_depth.z;
      this_node = node_idx + (x + (y << 1) + (z << 2));

      has_child = octree[2*this_node] & 0x40000000;

      if (has_child) {
        //The lowest 30 bits are the address
        node_idx = octree[2*this_node] & 0x3FFFFFFF;

        //Update the center depth for the next iteration
        center_depth.x += edge_length / 2 * (x ? 1 : -1);
        center_depth.y += edge_length / 2 * (y ? 1 : -1);
        center_depth.z += edge_length / 2 * (z ? 1 : -1);
      }
      edge_length /= 2.0f;
    }
    octree[2*this_node + 1] = values[index];
  }

}

__global__ void createCubeMeshFromSVO(int* octree, int* counter, int depth, float3 bbox0, float scale_factor, int num_voxels, float* cube_vbo,
  int cube_vbosize, int* cube_ibo, int cube_ibosize, float* cube_nbo, float* out_vbo, int* out_ibo, float* out_nbo, float* out_cbo) {

  //Get the index for the thread
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  float edge_length = abs(bbox0.x);
  float3 center = make_float3(0.0f, 0.0f, 0.0f);
  int pointer = 0;
  bool has_child = true;
  int val2;

  while (has_child) {
    //Get the lowest 3 bits to encode the first move
    int pos = idx & 0x7;

    //Get the value from the octree
    int val = octree[2*(pointer+pos)];
    val2 = octree[2*(pointer + pos) + 1];

    //It it is not occupied, do not continue
    has_child = val & 0x40000000;

    //Don't continue if it does not have a child
    if (!has_child && (octree[2 * (pointer + pos) + 1] >> 23 == 0)) {
      return;
    }

    //Get the child pointer for the next depth
    pointer = val & 0x3FFFFFFF;

    //Decode the value into xyz
    int x = pos & 0x1;
    int y = pos & 0x2;
    int z = pos & 0x4;

    //Update the center
    center.x += edge_length / 2 * (x ? 1 : -1);
    center.y += edge_length / 2 * (y ? 1 : -1);
    center.z += edge_length / 2 * (z ? 1 : -1);

    //Half the edge length for the next iteration
    edge_length /= 2.0f;

    //Shift right for the next iteration
    idx = idx >> 3;
  }

  int vidx = atomicAdd(counter, 1);
  //TODO: Detect if we exceed the allocated memory, and break out + allocate more

  if (vidx < num_voxels) {

    int vbo_offset = vidx * cube_vbosize;
    int ibo_offset = vidx * cube_ibosize;

    for (int i = 0; i < cube_vbosize; i++) {
      if (i % 3 == 0) {
        out_vbo[vbo_offset + i] = cube_vbo[i] * scale_factor + center.x;
        out_cbo[vbo_offset + i] = (float)((val2 & 0xFF) / 255.0);
      }
      else if (i % 3 == 1) {
        out_vbo[vbo_offset + i] = cube_vbo[i] * scale_factor + center.y;
        out_cbo[vbo_offset + i] = (float)(((val2 >> 7) & 0xFF) / 255.0);
      }
      else {
        out_vbo[vbo_offset + i] = cube_vbo[i] * scale_factor + center.z;
        out_cbo[vbo_offset + i] = (float)(((val2 >> 15) & 0xFF) / 255.0);
      }
      out_nbo[vbo_offset + i] = cube_nbo[i];
    }

    for (int i = 0; i < cube_ibosize; i++) {
      out_ibo[ibo_offset + i] = cube_ibo[i] + ibo_offset;
    }

  }

}

//This is based on Cyril Crassin's approach
__host__ void svoFromVoxels(int* d_voxels, int numVoxels, int* d_values, int* d_octree) {
  int numNodes = 8;
  int startingNode = 0;
  int* d_numNodes;
  hipMalloc((void**)&d_numNodes, sizeof(int));
  hipMemcpy(d_numNodes, &numNodes, sizeof(int), hipMemcpyHostToDevice);
  int depth = 0;

  while (numNodes < (numVoxels*log_N) && ++depth < log_N) {

    //First, parallelize on voxels and flag nodes to be subdivided
    flagNodes<<<(numVoxels / 256) + 1, 256>>>(d_voxels, numVoxels, d_octree, M, T, bbox0, t_d, p_d, depth);

    hipDeviceSynchronize();

    //Then, parallize on nodes and subdivide
    splitNodes<<<((numNodes - startingNode) / 256) + 1, 256>>>(&d_octree[2*startingNode], d_numNodes, numNodes - startingNode);
    startingNode = numNodes;

    hipDeviceSynchronize();
    hipMemcpy(&numNodes, d_numNodes, sizeof(int), hipMemcpyDeviceToHost);
  }

  //Now write values into the lowest level of the svo
  fillNodes<<<(numVoxels / 256) + 1, 256>>>(d_voxels, numVoxels, d_values, d_octree, M, T, bbox0, t_d, p_d);
  hipDeviceSynchronize();

  hipFree(d_numNodes);
}

__host__ void extractCubesFromSVO(int* d_octree, int numVoxels, Mesh &m_cube, Mesh &m_out) {

  //Move cube data to GPU
  thrust::device_vector<float> d_vbo_cube(m_cube.vbo, m_cube.vbo + m_cube.vbosize);
  thrust::device_vector<int> d_ibo_cube(m_cube.ibo, m_cube.ibo + m_cube.ibosize);
  thrust::device_vector<float> d_nbo_cube(m_cube.nbo, m_cube.nbo + m_cube.nbosize);

  //Create output structs
  float* d_vbo_out;
  int* d_ibo_out;
  float* d_nbo_out;
  float* d_cbo_out;
  hipMalloc((void**)&d_vbo_out, numVoxels * m_cube.vbosize * sizeof(float));
  hipMalloc((void**)&d_ibo_out, numVoxels * m_cube.ibosize * sizeof(int));
  hipMalloc((void**)&d_nbo_out, numVoxels * m_cube.nbosize * sizeof(float));
  hipMalloc((void**)&d_cbo_out, numVoxels * m_cube.nbosize * sizeof(float));

  //Warn if vbo and nbo are not same size on cube
  if (m_cube.vbosize != m_cube.nbosize) {
    std::cout << "ERROR: cube vbo and nbo have different sizes." << std::endl;
    return;
  }

  //Create global counter to determine where to write the output
  int* d_counter;
  int initial_count = 0;
  hipMalloc((void**)&d_counter, sizeof(int));
  hipMemcpy(d_counter, &initial_count, sizeof(int), hipMemcpyHostToDevice);

  //Create resulting cube-ized mesh
  createCubeMeshFromSVO << <(N*N*N / 256) + 1, 256 >> >(d_octree, d_counter, log_N, bbox0, vox_size / CUBE_MESH_SCALE, numVoxels, thrust::raw_pointer_cast(&d_vbo_cube.front()),
    m_cube.vbosize, thrust::raw_pointer_cast(&d_ibo_cube.front()), m_cube.ibosize, thrust::raw_pointer_cast(&d_nbo_cube.front()), d_vbo_out, d_ibo_out, d_nbo_out, d_cbo_out);

  //Store output sizes
  m_out.vbosize = numVoxels * m_cube.vbosize;
  m_out.ibosize = numVoxels * m_cube.ibosize;
  m_out.nbosize = numVoxels * m_cube.nbosize;
  m_out.cbosize = m_out.nbosize;

  //Memory allocation for the outputs
  m_out.vbo = (float*)malloc(m_out.vbosize * sizeof(float));
  m_out.ibo = (int*)malloc(m_out.ibosize * sizeof(int));
  m_out.nbo = (float*)malloc(m_out.nbosize * sizeof(float));
  m_out.cbo = (float*)malloc(m_out.cbosize * sizeof(float));

  //Sync here after doing some CPU work
  hipDeviceSynchronize();

  //Copy data back from GPU
  //TODO: Can we avoid this step by making everything run from device-side VBO/IBO/NBO/CBO?
  hipMemcpy(m_out.vbo, d_vbo_out, m_out.vbosize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(m_out.ibo, d_ibo_out, m_out.ibosize*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(m_out.nbo, d_nbo_out, m_out.nbosize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(m_out.cbo, d_cbo_out, m_out.cbosize*sizeof(float), hipMemcpyDeviceToHost);

  ///Free GPU memory
  hipFree(d_vbo_out);
  hipFree(d_ibo_out);
  hipFree(d_nbo_out);
  hipFree(d_counter);
}

__host__ void voxelizeSVOCubes(Mesh &m_in, Mesh &m_cube, Mesh &m_out) {

  //Voxelize the mesh input
  int numVoxels = N*N*N;
  int* d_voxels;
  int* d_values;
  hipMalloc((void**)&d_voxels, numVoxels*sizeof(int));
  hipMalloc((void**)&d_values, numVoxels*sizeof(int));
  numVoxels = voxelizeMesh(m_in, d_voxels, d_values);

  //Create the octree
  int* d_octree = NULL;
  hipMalloc((void**)&d_octree, 32*log_N*numVoxels*sizeof(int));
  svoFromVoxels(d_voxels, numVoxels, d_values, d_octree);

  //Extract cubes from the leaves of the octree
  extractCubesFromSVO(d_octree, numVoxels, m_cube, m_out);

  //Free up GPU memory
  hipFree(d_voxels);
  hipFree(d_octree);

}
